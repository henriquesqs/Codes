#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void dotProduct(int *a, int *b, int *dot_global, int n) {
	
	int tid = threadIdx.x; // identificador local da thread
	int localDot = 0, aux = 0;
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < n) {
		for(int i = 0; i < n; i++) {
			aux = a[tid * n + i] * b[tid * n + i];
			localDot += aux;
		}
	}

	atomicAdd(dot_global, localDot);
}

__global__ void minValue(int *a, int *b, int *min_global, int n) {
	
	int tid = threadIdx.x; // identificador local da thread
	int localMin = INT_MAX, aux = 0;
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id < n) {
		for (int i = 0; i < n; i++) {
			aux = min(a[tid * n + i], b[tid * n + i]);
			localMin = min(localMin, aux);
		}
	}
	
	atomicMin(min_global, localMin);
}

__global__ void maxValue(int *a, int *b, int *max_global, int n) {

	int tid = threadIdx.x; // identificador local da thread
	int localMax = INT_MIN, aux = 0;
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id < n) {
		for (int i = 0; i < n; i++) {
			aux = max(a[tid * n + i], b[tid * n + i]);
			localMax = max(localMax, aux);
		}
	}

	atomicMax(max_global, localMax);
}

int main(int argc, char **argv) {

	// Verificando se o usuário inseriu os argumentos corretamente
	if (argc < 2) {
		printf("Missing filename argument.\n");
		return 0;
	}
	
	// Lendo arquivo de entrada
	FILE *pFile;
	pFile = fopen(argv[1], "r");

	if (pFile == NULL) {
		printf("Error opening file.\n");
		return 0;
	}

	int n; // Tamanho das matrizes
	int *a, *b; // Matrizes
	int numBlocks; // Numero de blocos por grid
	int threadsPerBlock; // Numero de threads por bloco
	int *min, *max, *dot; // Variaveis de resultado
	hipStream_t stream1, stream2, stream3; // Variáveis pra criação de streams

	// Criação das streams.
	// Cada uma ficará responsável por uma operação solicitada:
	// Stream1: produto escalar; Stream2: maior valor; Stream3: menor valor
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipStreamCreate(&stream3);

	// Lendo tamanho das matrizes
	// scanf("%d\n", &n);
	fscanf(pFile, "%d\n", &n);

	// Aloca as matrizes 'a', 'b' no host e no device
	// (memória mapeada (pinned memory) em endereço virtual unificado)
	// É mais rapido do que usar malloc convencional
	hipHostAlloc((void **)&a, n * n * (sizeof(int)), hipHostMallocMapped);
	hipHostAlloc((void **)&b, n * n * (sizeof(int)), hipHostMallocMapped);

	// Aloca as variáveis de resultado no host e no device
	hipHostAlloc((void **)&min, sizeof(int), hipHostMallocMapped);
	hipHostAlloc((void **)&max, sizeof(int), hipHostMallocMapped);
	hipHostAlloc((void **)&dot, sizeof(int), hipHostMallocMapped);

	// Lendo matriz 'a'
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			// scanf("%d ", &a[i * n + j]);
			fscanf(pFile, "%d ", &a[i * n + j]);
		}
	}

	// Exibindo a matriz 'a'
	// printf("Matriz a:\n");
	// for (int i = 0; i < n; i++){
	// 	for (int j = 0; j < n; j++){
	// 		printf("%d ", a[i * n + j]);
	// 	}
	// 	printf("\n");
	// }

	// Lendo a matriz 'b'
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			// scanf("%d ", &b[i * n + j]);
			fscanf(pFile, "%d ", &b[i * n + j]);
		}
	}

	// Exibindo a matriz 'b'
	// printf("\nMatriz b:\n");
	// for (int i = 0; i < n; i++){
	// 	for (int j = 0; j < n; j++){
	// 		printf("%d ", b[i * n + j]);
	// 	}
	// 	printf("\n");
	// }

	numBlocks = 256;
	threadsPerBlock = (numBlocks/n)+1;
	
	*min = INT_MAX; // inicializa min global
	*max = INT_MIN; // inicializa max global

	dotProduct<<<numBlocks, threadsPerBlock, 0, stream1>>>(a, b, dot, n);
	minValue<<<numBlocks, threadsPerBlock, 0, stream2>>>(a, b, min, n);
	maxValue<<<numBlocks, threadsPerBlock, 0, stream3>>>(a, b, max, n);
	
	// Sincroniza pois não teremos a função cudaMemcpy
	// que tem a sincronização implicita
	hipDeviceSynchronize();
	
	// Exibindo resultados
	printf("%d %d %d\n", *dot, *min, *max);

	// Liberando memória das matrizes e das variáveis de resultado
	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(min);
	hipHostFree(max);
	hipHostFree(dot);

	// Destroi as streams
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);
	hipStreamDestroy(stream3);

	return 0;
}