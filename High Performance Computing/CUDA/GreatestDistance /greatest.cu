/*	
	
	Grupo 13 - Turma B
	
	Gustavo T. Mastrobuono, 			NUSP 10734411;
	Henrique de S. Q. dos Santos, NUSP 10819029;
	Jhordan P. V. Pesantes, 			NUSP 11733353;
	Witor M. A. de Oliveira, 			NUSP 10692190;
	Yorvin A. R. Carrion, 				NUSP 11733332;

*/

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define THREADS 4

// Função original por Paulo S L de Souza
// Modificada pelo grupo
__global__ void calc_diferenca(int *matriz, int *max_diff, int *maior, int *menor, int n){

	int i, j;
	int dif = INT_MIN, aux_dif = INT_MIN; // maior local, diferenca atual

	int linha = blockIdx.x * blockDim.x + threadIdx.x;
	int coluna = blockIdx.y * blockDim.y + threadIdx.y;
	
	for (i = (linha - 1); i < (linha + 2); i++){
		for (j = (coluna - 1); j < (coluna + 2); j++){

			if( (i < n && i >= 0) && (j < n && j >= 0) && (linha < n && linha >= 0) && (coluna < n && coluna >= 0) ){
			
				aux_dif = abs(matriz[linha * n + coluna] - matriz[i * n + j]);

				if(aux_dif > dif){

					dif = aux_dif;
					
					atomicMax(max_diff, dif);
					
					if(dif == *max_diff){
						
						if (matriz[linha * n + coluna] > matriz[i * n + j]){
							atomicExch(maior, matriz[linha * n + coluna]);
							atomicExch(menor, matriz[i * n + j]);
						}
						
						else {
							atomicExch(maior, matriz[i * n + j]);
							atomicExch(menor, matriz[linha * n + coluna]);
						}
					}
				}
			}
		}
	}
}

int main(int argc, char **argv){

	int n; // tamanho da matriz
	int *a; // matriz
	int *max_diff, *maior, *menor; // variaveis de resultado

	// Lendo tamanho das matrizes
	scanf("%d\n", &n);

	// Aloca a matriz 'a' no host e no device
	// (memória mapeada (pinned memory) em endereço virtual unificado)
	// É mais rapido do que usar malloc convencional.
	hipHostAlloc((void **)&a, n * n * (sizeof(int)), hipHostMallocMapped);

	// Aloca as variáveis de resultado no host e no device
	hipHostAlloc((void **)&max_diff, sizeof(int), hipHostMallocMapped);
	hipHostAlloc((void **)&maior, sizeof(int), hipHostMallocMapped);
	hipHostAlloc((void **)&menor, sizeof(int), hipHostMallocMapped);

	// Lendo matriz 'a'
	for (int i = 0; i < n; i++){
		for (int j = 0; j < n; j++){
			scanf("%d ", &a[i * n + j]);
		}
	}

	// for (int i = 0; i < n; i++){
	// 	for (int j = 0; j < n; j++){
	// 		printf("%d ", a[i * n + j]);
	// 	}
	// 	printf("\n");
	// }

	*max_diff = INT_MIN; // inicializa max global

	// Define a quantidade de threads por bloco
	dim3 threadsPerBlock(THREADS,THREADS);

	// Define a quantidade de blocos por grade
	dim3 blocksPerGrid(((n+(threadsPerBlock.x-1)) / threadsPerBlock.x), (n+(threadsPerBlock.y-1)) / threadsPerBlock.y);

	calc_diferenca<<<blocksPerGrid, threadsPerBlock>>>(a, max_diff, maior, menor, n);

	// Sincroniza pois não teremos a função cudaMemcpy
	// que tem a sincronização implicita
	hipDeviceSynchronize();

	// Exibindo resultados
	printf("%d %d \n", *maior, *menor);

	hipHostFree(a);
	hipHostFree(max_diff);
	hipHostFree(maior);
	hipHostFree(menor);

	return 0;
}
