#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void calculaLimiar(int *mat, int lin, int col, int limiar) {

	int i = blockDim.x * blockIdx.x + threadIdx.x; 
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if(i < lin && j < col){
		if(mat[i * col + j] > limiar){
			mat[i * col + j] = 1;
		}
		else{
			mat[i * col + j] = 0;
		}
	}

}

int main(int argc, char **argv) {

	// Verificando se o usuário inseriu os argumentos corretamente
	if (argc < 2) {
		printf("Missing filename argument.\n");
		return 0;
	}
	
	// Lendo arquivo de entrada
	FILE *pFile;
	pFile = fopen(argv[1], "r");

	if(pFile == NULL) {
		printf("Error opening file.\n");
		return 0;
	}
	
	int *mat; // matriz
	int limiar; // limiar da matriz
	int lin, col; // dimensoes da matriz
	int blockSize = 32; // tamanho do bloco dentro do grid

	// Lendo tamanho e limiar da matriz
	fscanf(pFile, "%d %d\n%d\n", &lin, &col, &limiar);

	// Alocação da matriz (pinned memory, pois é mais rápida)
	// A matriz já está sendo alocada no host e no device
	hipHostAlloc((void **)&mat, lin * col * (sizeof(int)), hipHostMallocMapped);

	// Leitura dos dados da matriz
	for(int i = 0; i < lin; i++){
		for(int j = 0; j < col; j++){
			fscanf(pFile, "%d ", &mat[i * col + j]);
		}
	}

	// Exibição dos dados da matriz
	printf("Matriz lida:\n");
	for(int i = 0; i < lin; i++){
		for(int j = 0; j < col; j++){
			printf("%d ", mat[i * col + j]);
		}
		printf("\n");
	}

	dim3 threadsPerBlock(blockSize, blockSize);
	dim3 numBlocks(ceil(col/(float)threadsPerBlock.x), ceil(lin/(float)threadsPerBlock.y));

  	calculaLimiar<<<numBlocks, threadsPerBlock>>>(mat, lin, col, limiar);

	// Sincroniza pois não tem o cudaMemcpy que tem a sincronização implicita
	hipDeviceSynchronize();

	// Exibição dos dados da matriz após aplicação do limiar
	printf("\nMatriz resultante:\n");
	for(int i = 0; i < lin; i++){
		for(int j = 0; j < col; j++){
			printf("%d ", mat[i * col + j]);
		}
		printf("\n");
	}

  	hipHostFree(mat);

	return 0;
}