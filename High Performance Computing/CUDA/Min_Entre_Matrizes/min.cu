#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void min(int* mat, int lin, int col, int* min_global){
	
	int min_local = INT_MAX;
	
	int i = blockDim.x * blockIdx.x + threadIdx.x; 
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	
	if(i < lin && j < col) {
		if(mat[i * col + j] < *min_global){
			min_local = mat[i * col + j];
			atomicMin(min_global, min_local);
		}
	}

}

int main(int argc, char **argv) {

	// Verificando se o usuário inseriu os argumentos corretamente
	if (argc < 2) {
		printf("Missing filename argument.\n");
		return 0;
	}
	
	// Lendo arquivo de entrada
	FILE *pFile;
	pFile = fopen(argv[1], "r");

	if(pFile == NULL) {
		printf("Error opening file.\n");
		return 0;
	}
	
	int *a, *b; // matrizes
	int blockSize = 16; // tamanho do bloco dentro do grid
	int *min_a, *min_b; // variavel de resultado
	int lin_a, col_a, lin_b, col_b; // dimensoes das matrizes
    hipStream_t stream1,stream2; // variaveis de stream

    //Criação das streams
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

	// Lendo tamanho das matrizes
	fscanf(pFile, "%d %d\n%d %d\n", &lin_a, &col_a, &lin_b, &col_b);

	// Alocação da matriz 'a' e 'b' (pinned memory, pois é mais rápida)
	// A matriz já está sendo alocada no host e no device
	hipHostAlloc((void **)&a, lin_a * col_a * (sizeof(int)), hipHostMallocMapped);
	hipHostAlloc((void **)&b, lin_b * col_b * (sizeof(int)), hipHostMallocMapped);

	// Alocação das variaveis de resultado no host e no device
	hipHostAlloc((void **)&min_a, (sizeof(int)), hipHostMallocMapped);
	hipHostAlloc((void **)&min_b, (sizeof(int)), hipHostMallocMapped);

	// Lendo elementos da matriz 'a'
	for(int i = 0; i < lin_a; i++){
		for(int j = 0; j < col_a; j++){
			fscanf(pFile, "%d ", &a[i * col_a + j]);
		}
	}

	// Exibindo elementos da matriz 'a'
	printf("Matriz a:\n");
	for(int i = 0; i < lin_a; i++){
		for(int j = 0; j < col_a; j++){
			printf("%d ", a[i * col_a + j]);
		}
		printf("\n");
	}

	// Lendo elementos da matriz 'b'
	for(int i = 0; i < lin_b; i++){
		for(int j = 0; j < col_b; j++){
			fscanf(pFile, "%d ", &b[i * col_b + j]);
		}
	}

	// Exibindo elementos da matriz 'b'
	printf("\nMatriz b:\n");
	for(int i = 0; i < lin_b; i++){
		for(int j = 0; j < col_b; j++){
			printf("%d ", b[i * col_b + j]);
		}
		printf("\n");
	}

	dim3 numBlocksA(ceil(lin_a/(float)blockSize), ceil(col_a/(float)blockSize));
	dim3 threadsPerBlockA(blockSize, blockSize);

	dim3 numBlocksB(ceil(lin_b/(float)blockSize), ceil(col_b/(float)blockSize));
	dim3 threadsPerBlockB(blockSize, blockSize);

	// Inicializando variaveis de min_global
	*min_a = INT_MAX;
	*min_b = INT_MAX;

	min<<<numBlocksA, threadsPerBlockA,0,stream1>>>(a, lin_a, col_a, min_a);
	min<<<numBlocksB, threadsPerBlockB,0,stream2>>>(b, lin_b, col_b, min_b);

	// Sincroniza pois não teremos a função cudaMemcpy
	// que tem a sincronização implicita
	hipDeviceSynchronize();

	printf("\nResultado:\n%d\n%d\n", *min_a, *min_b);

	// Libera memória das matrizes e das variaveis de resultado
	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(min_a);
	hipHostFree(min_b);

	// Destroi as streams
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

	return 0;
}