/********************************************************************/
/* Naive NxM matriz multiplication using CUDA without shared memory */
/********************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

__device__ void variables(){

	int indice_local = threadIdx.x;
	int indice_global = blockDim.x * blockIdx.x + threadIdx.x;
	int indice_bloco = blockIdx.x;
	int tamanho_grid = gridDim.x;

	printf("Hello from thread: índice global %d, índice local: %d, bloco: %d, tamanho do grid: %d\n\n", indice_global, indice_local, indice_bloco, tamanho_grid);
}

// 
// Found this function here:
// https://medium.com/analytics-vidhya/matrix-multiplication-in-cuda-a-simple-guide-bab44bc1f8ab
// 
// Probably it is based on chapter 3.2.4.Shared Memory from
// https://docs.nvidia.com/cuda/pdf/CUDA_C_Programming_Guide.pdf
// 
__global__ void mul(int *a, int *b, int *c, int n, int m, int width) {

	// Each thread computes one element of C
	// by accumulating results into 'sum'
  int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;   
	
	// check boundry conditions
  if(row < n && col < m){
		
		// variables();		

		int sum = 0;

    // do the multiplication for one row and col
    for(int k = 0; k < width; k++){
			sum += a[row * width + k] * b[k * m + col];
		}

    // store the result
    c[row * m + col] = sum;
  }
	
}

int main(int argc, char **argv){	

	int lin_a, col_a, col_b; // qntd de linhas e colunas das matrizes
	int *a, *b, *c; // matrizes do host
	int *dev_a, *dev_b, *dev_c; // matrize do device

	// Lendo numero de linhas e colunas da matriz A e B
	scanf("%d %d %d\n", &lin_a, &col_a, &col_b);

	// Alocando as cópias das matrizes no host
	a = (int*) malloc(lin_a * col_a * sizeof(int));
	b = (int*) malloc(col_a * col_b * sizeof(int));
	c = (int*) malloc(lin_a * col_b * sizeof(int));

	// Alocando as cópias das matrizes no device
	hipMalloc((void**)&dev_a, lin_a * col_a * sizeof(int));
	hipMalloc((void**)&dev_b, col_a * col_b * sizeof(int));
	hipMalloc((void**)&dev_c, lin_a * col_b * sizeof(int));
	
	// Lendo valores da matriz "a"
	for(int i = 0; i < lin_a; i++){
		for(int j = 0; j < col_a; j++){
			scanf("%d ", &a[i * col_a + j]);
		}
	}

	// Exibindo valores da matriz "a"
	printf("\nMatriz a:\n");
	for(int i = 0; i < lin_a; i++){
		for(int j = 0; j < col_a; j++){
			printf("%d ", a[i * col_a + j]);
		}
		printf("\n");
	}

	// Lendo valores da matriz "b"
	for(int i = 0; i < col_a; i++){
		for(int j = 0; j < col_b; j++){
			scanf("%d ", &b[i * col_b + j]);
		}
	}

	// Exibindo valores da matriz "b"
	printf("\nMatriz b:\n");
	for(int i = 0; i < col_a; i++){
		for(int j = 0; j < col_b; j++){
			c[i * col_b + j] = 0;
			printf("%d ", b[i * col_b + j]);
		}
		printf("\n");
	}
	printf("\n");

	// Copiando valores de 'a', 'b' e 'c' em suas cópias no device
	hipMemcpy(dev_a, a, lin_a * col_a * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, col_a * col_b * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, lin_a * col_b * sizeof(int), hipMemcpyHostToDevice);

	dim3 numBlocks(ceil(lin_a/(float)BLOCK_SIZE), ceil(col_b/(float)BLOCK_SIZE));
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);

	mul<<<numBlocks,threadsPerBlock>>>(dev_a, dev_b, dev_c, lin_a, col_b, col_a);

	// Copiando a matriz 'c' do device pro host. 
	// Essa função tem um CudaDevSync implícito.
	hipMemcpy(c, dev_c, lin_a * col_b * sizeof(int), hipMemcpyDeviceToHost);

	// Exibindo valores da matriz "c"
	printf("Matriz c:\n");
	for(int i = 0; i < lin_a; i++){
		for(int j = 0; j < col_b; j++){
			printf("%d ", c[i * col_b + j]);
		}
		printf("\n");
	}

	// Liberando memória
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	free(a);
	free(b);
	free(c);
  
	return 0;
}